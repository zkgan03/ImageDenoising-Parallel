
#include "CudaWrapper.cuh"

namespace CudaWrapper {

    // Memory management
    void malloc(void** devPtr, size_t size) {
        CUDA_CALL(hipMalloc(devPtr, size));
    }

    void free(void* devPtr) {
        CUDA_CALL(hipFree(devPtr));
    }

    void hostAllocate(void** hostPtr, size_t size, unsigned int flags) {
        CUDA_CALL(hipHostAlloc(hostPtr, size, flags));
    }

    void hostFree(void* hostPtr) {
        CUDA_CALL(hipHostFree(hostPtr));
    }

    // Stream management
    void streamCreate(hipStream_t* stream) {
        CUDA_CALL(hipStreamCreate(stream));
    }

    void streamDestroy(hipStream_t stream) {
        CUDA_CALL(hipStreamDestroy(stream));
    }

    void streamSynchronize(hipStream_t stream) {
        CUDA_CALL(hipStreamSynchronize(stream));
    }

    // Memory transfer
    void memcpy(void* dst, const void* src, size_t size, hipMemcpyKind kind, hipStream_t stream) {
        if (stream) {
            CUDA_CALL(hipMemcpyAsync(dst, src, size, kind, stream));
        }
        else {
            CUDA_CALL(hipMemcpy(dst, src, size, kind));
        }
    }

} // namespace CudaWrapper
